#include "hip/hip_runtime.h"
//
// Created by xingyu on 5/24/23.
//
#include "core/cuda_utils.h"
#include "core/utils.h"
#include "arap.h"


#include <iostream>

//typedef Eigen::MatrixXd<float, Eigen::Dynamic, Eigen::Dynamic, Eigen::Dynamic> Matrix3f;

void save_mesh(at::Tensor xyz, at::Tensor faces, std::string filename) {
    std::ofstream file(filename);
    defer {file.close(); };
    file << "OFF\n";
    const int N = xyz.size(0);
    const int M = faces.size(0);

    file << N << " " << M << " " << 0 << "\n";
    for (int i = 0; i < N; ++i) {
        file << xyz[i][0].item<float>() << " " << xyz[i][1].item<float>() << " " << xyz[i][2].item<float>() << "\n";
    }
    for (int i = 0; i < M; ++i) {
        file << "3 " << faces[i][0].item<int>() << " " << faces[i][1].item<int>() << " " << faces[i][2].item<int>()<< "\n";
    }
}

int main() {
    std::string filename = "/data_HDD/zhuxingyu/vscode/py3d/data/decimated_knight.off";
//    std::string filename = "/data_HDD/zhuxingyu/.dataset/ModelNet40/door/test/door_0110.off";
    std::ifstream mesh_file(filename);
    defer {mesh_file.close(); };

//    mesh_file.ignore(1);
    std::string head;
    mesh_file >> head;
    int V, F, VN;
    mesh_file >> V >> F >> VN;
    std::cout << V << " " << F << " " << VN << std::endl;
    std::vector<float> vs(V * 3);
    std::vector<int> fs(F * 3);
    for (int i = 0; i < V; ++i) {
        float a, b, c;
        mesh_file >> a >> b >> c;
        vs[3*i] = a;
        vs[3*i+1] = b;
        vs[3*i+2] = c;
    }
    for (int i = 0; i < F; ++i) {
        int n, a, b, c;

        mesh_file >> n >> a >> b >> c;
        fs[3*i] = a;
        fs[3*i+1] = b;
        fs[3*i+2] = c;
    }
    std::cout << vs.size() << " " << fs.size() << std::endl;

    at::Tensor xyz = torch::from_blob(vs.data(), {1, V, 3}, at::device(at::kCPU).dtype(at::ScalarType::Float)).contiguous();
    at::Tensor faces = torch::from_blob(fs.data(), {1, F, 3}, at::device(at::kCPU).dtype(at::ScalarType::Int));
    at::Tensor move = torch::randn({1, 3}, at::device(at::kCPU).dtype(at::ScalarType::Float));
    at::Tensor handle = torch::randint(V, {1, 5}, at::device(at::kCPU).dtype(at::ScalarType::Int));

    auto max_values = std::get<0>(xyz.max(1, true));
    auto min_values = std::get<0>(xyz.min(1, true));
    auto ranges = max_values - min_values;
    xyz = (xyz - min_values) / ranges;

    move = move / move.norm(2, -1, true);
    DEBUG(move);

    at::Tensor new_xyz = arap_deform(xyz, faces, handle, move);
    DEBUG((new_xyz - xyz).sum());
    new_xyz = (new_xyz + min_values) * ranges;
    save_mesh(new_xyz[0], faces[0], "/data_HDD/zhuxingyu/vscode/py3d/data/out.off");
    DEBUG("heyehye");
    return 0;
}